#include "hip/hip_runtime.h"
#include <tokenizer/tokenizer.cuh>

namespace wasabi::gpu {
void GPUTokenizer::Tokenize() {
  const std::string corpus_raw = this->corpus_->corpus_blob();
}

__global__ void GPUTokenizer::kTokenizeToWords(const char* corpus,
                                               char* outputs) {
  int block_row = blockIdx.y;
  int block_col = blockIdx.x;

  // Splice into threads of the char array
}
}  // namespace wasabi::gpu
}  // namespace wasabi::gpu